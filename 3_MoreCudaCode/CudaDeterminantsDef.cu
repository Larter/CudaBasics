#include <hip/hip_runtime.h>


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#define VALUESMAX 100
#define BMARK -1
#define SIZE 2
#define PRINTMATRIX 1
#define PRINTPERM 0
#define SINGLETONS 0 //1 if singletons 0 if inversions
#define RAND 5

__device__ void nextPermutationBlock(long double *matrix, long double *permutations, bool *usedValues, int n, long double value, int parametr, int *fractionNumber, int level);
__global__ void permutations(long double *matrix,long double *permutationValues);
__global__ void addPermutations(long double *determinant, long double *permutations, int *n);

int main(){

	long double *matrix, *d_matrix, *d_permutationValues, *d_determinant, determinant;
	int n=SIZE, *d_n;
	hipError_t cudaStatus;

	matrix=(long double*)malloc(sizeof(long double)*n*n);//alocating matrix

	for(int i=0; i< n*n; i++){
		matrix[i]=rand()%RAND;
		if(PRINTMATRIX==1){
			printf("%Lf ",matrix[i]);
			if(!((i+1)%n)){
			    printf("\n");
			}//if
		}//PRINTMATRIX
	}//for


	cudaStatus=hipMalloc((void**)&d_matrix, n*n*sizeof(long double)); /*allocating matrix memory on gpu*/

	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"Error in allocating memory\n");
		return hipSuccess;
	}

	cudaStatus=hipMalloc((void**)&d_n,sizeof(int));

	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"Error in allocating memory\n");
		return cudaStatus;
	}


	cudaStatus=hipMalloc((void**)&d_permutationValues, (n-1)*n*sizeof(long double)); /*allocating matrix memory on gpu*/

	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"Error in allocating memory\n");
		return hipSuccess;
	}

	cudaStatus=hipMalloc((void**)&d_determinant, sizeof(long double)); /*allocating matrix memory on gpu*/

	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"Error in allocating memory\n");
		return hipSuccess;
	}


	cudaStatus=hipMemcpy(d_matrix,matrix,n*n*sizeof(long double),hipMemcpyHostToDevice);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"Error in copying matrix memory, %d\n");
		return cudaStatus;
	}

	cudaStatus=hipMemcpy(d_n,&n,sizeof(int),hipMemcpyHostToDevice);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"Error in copying matrix memory\n");
		return cudaStatus;
	}









	permutations<<<n,n-1>>>(d_matrix,d_permutationValues);


	//copying memory

	addPermutations<<<1,1>>>(d_determinant,d_permutationValues,d_n);

	cudaStatus=hipMemcpy(&determinant,d_determinant,sizeof(long double),hipMemcpyDeviceToHost);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"Error in copying matrix memory, %d\n");
		return cudaStatus;
	}

	printf("Determinant equals: %Lf \n", determinant);


}




__device__ void nextPermutationBlock(long double *matrix, long double *permutations, bool *usedValues, int n, long double value, int parametr, int *fractionNumber, int level){


	if(level==n){
		if(SINGLETONS==1){
			value*=pow((float)BMARK,n-parametr);
		}
		else{
			value*=pow((float)BMARK,parametr);
		}
		*permutations+=value;

	}
	else{
		int addValue=-1;
		for(int i=0; i<n; i++){
			if(usedValues[i]==true){
				usedValues[i]=false;
				if(SINGLETONS==1){
					if(i==level){
						addValue=1;
					}
					else{
						addValue=0;
					}
				}
				else{//inversions
					addValue++;
				}


				nextPermutationBlock(matrix,permutations,usedValues,n,value*matrix[level*n+i],parametr+addValue,fractionNumber,level+1);
				usedValues[i]=true;
			}


		}

	}
}


__global__ void addPermutations(long double *determinant, long double *permutations, int *n){

	int nn=*n**n-1;
	*determinant=0;
	for(int i=0;i<nn;i++){
		*determinant+=permutations[i];
	}
}

__global__ void permutations(long double *matrix,long double *permutationValues){

	int *fractionNumber[1]={0};
	int n=gridDim.x;
	int inversions;
	bool usedValues[VALUESMAX];
	double result=1;
	for(int i=0; i<blockDim.x+1;i++){
		usedValues[i]=true;
	}
	usedValues[blockIdx.x]=false;
	usedValues[threadIdx.x]=false;
	result*=matrix[blockIdx.x];
	result*=matrix[n+threadIdx.x];

	inversions=blockIdx.x+threadIdx.x;
	if(blockIdx.x<threadIdx.x){
		inversions--;
	}

	nextPermutationBlock(matrix, &permutationValues[blockIdx.x*n+threadIdx.x],usedValues,n,result,inversions,fractionNumber[0], 2);


}
