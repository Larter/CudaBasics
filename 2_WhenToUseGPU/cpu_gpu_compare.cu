#include <iostream>
#include <iterator>
#include <algorithm>
#include <hip/hip_runtime.h>


#include <sys/times.h>
#include <stdint.h>


void start_clock(void);
void end_clock(char *msg);
static clock_t st_time;
static clock_t en_time;
static struct tms st_cpu;
static struct tms en_cpu;

void
start_clock()
{
    st_time = times(&st_cpu);
}


void end_clock(char *msg)
{
    en_time = times(&en_cpu);

    std::cout<< "Time on " << msg << " :"<< (intmax_t)(en_time - st_time)<<std::endl;
}

void pow_array(float *a, int power, int array_size)
{
for(int idx =0; idx<array_size; ++idx)
{
  float result=1;
  if (idx<array_size)
  {
      for(int i=0; i<power; ++i)
        result*=a[idx];
    a[idx] = result;
  }
}
}

// Kernel that executes on the CUDA device
__global__ void pow_array_gpu(float *a, int power, int array_size)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  float result=1;
  if (idx<array_size)
  {
      for(int i=0; i<power; ++i)
        result*=a[idx];
    a[idx] = result;
  }
}


int main(int argc, char ** argv)
{

if(argc<3)
{
  std::cout<<"Please provide array size, and iteration level as arguments"<<std::endl;
  return 1;
}
{
  start_clock();
  int array_size = atoi(argv[1]);
  int power = atoi(argv[2]);

  float *a_host = new float[array_size]; //array on CPU
  for (int i=0; i<array_size; i++) a_host[i] = i;


  float *a_device;

//  cudaDeviceReset();

  long mallocStart= clock();
  hipMalloc((void **) &a_device, array_size*sizeof(float));   // Allocate array on device

  long memcpyStart=clock();
  hipMemcpy(a_device, a_host, array_size*sizeof(float), hipMemcpyHostToDevice);



  // Do calculation on device:
  int block_size = 256;

  int n_blocks = array_size/block_size + (array_size%block_size == 0 ? 0:1);

  pow_array_gpu <<< n_blocks, block_size >>> (a_device, power, array_size);
  // Retrieve result from device and store it in host array


  hipMemcpy(a_host, a_device, sizeof(float)*array_size, hipMemcpyDeviceToHost);

  // Print results
  std::ostream_iterator<float> printIterator(std::cout, "\n");
  std::copy(a_host, a_host+2, printIterator);
  std::cout<<"."<<std::endl;
  std::copy(a_host+array_size-2, a_host+array_size, printIterator);

  // Cleanup
  free(a_host); hipFree(a_device);

  end_clock("GPU");
  }
  {
  start_clock();
  int array_size = atoi(argv[1]);
  int power = atoi(argv[2]);

  float *a_host = new float[array_size]; //array on CPU
  for (int i=0; i<array_size; i++) a_host[i] = i;



  pow_array(a_host, power, array_size);



  // Print results
  std::ostream_iterator<float> printIterator(std::cout, "\n");
  std::copy(a_host, a_host+2, printIterator);
  std::cout<<"."<<std::endl;
  std::copy(a_host+array_size-2, a_host+array_size, printIterator);

  // Cleanup
  free(a_host);
  end_clock("CPU");
  }
}